
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void firstParallel()
{
  printf("This is running in parallel.\n");
}

int main()
{
  firstParallel<<<4, 4>>>();
  hipDeviceSynchronize();
}


#include <hip/hip_runtime.h>
#include <stdio.h>

// Define a custom data transfer function from host to device
void customCopyHostToDevice(int* dst, const int* src, size_t size)
{
    for (size_t i = 0; i < size; i++)
    {
        dst[i] = src[i];
    }
}

// Define a custom data transfer function from device to host
void customCopyDeviceToHost(int* dst, const int* src, size_t size)
{
    for (size_t i = 0; i < size; i++)
    {
        dst[i] = src[i];
    }
}

/*
 * Initialize array values on the host.
 */

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

/*
 * Double elements in parallel on the GPU.
 */

__global__
void doubleElements(int *a, int N)
{
  int i;
  i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N)
  {
    a[i] *= 2;
  }
}

/*
 * Check all elements have been doubled on the host.
 */

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i * 2) return false;
  }
  return true;
}

int main()
{
  int N = 100;
  int *a;
  int *d_a; // Device pointer

  size_t size = N * sizeof(int);

  /*
   * Allocate memory for `a` on the host.
   */

  a = (int *)malloc(size);

  init(a, N);

  size_t threads_per_block = 10;
  size_t number_of_blocks = 10;

  /*
   * Allocate memory for `d_a` on the device.
   * Note: No cudaMalloc used; memory is statically allocated.
   */

  d_a = (int *)malloc(size);

  /*
   * Custom data transfer from host to device.
   */

  customCopyHostToDevice(d_a, a, size);

  /*
   * This launch will work because the pointer `d_a` is available to the device.
   */

  doubleElements<<<number_of_blocks, threads_per_block>>>(d_a, N);
  hipDeviceSynchronize();

  /*
   * Custom data transfer from device to host.
   */

  customCopyDeviceToHost(a, d_a, size);

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  /*
   * Free allocated memory on both the host and the device.
   */

  free(a);
  free(d_a);
}
